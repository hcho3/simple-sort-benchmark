
#include <hip/hip_runtime.h>
#include <cstdio>

int main(void)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    printf("-gencode arch=compute_%d%d,code=sm_%d%d\n",
        prop.major, prop.minor, prop.major, prop.minor);

    return 0;
}
